#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <iomanip>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#define N 100
#define numThreads 512


__global__ void initialize(
	unsigned int seed, 
	hiprandState_t *states,
	unsigned int size)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < size)
	{
		hiprand_init(
			seed,
			i,
			0,
			&states[i]);
	}
}


__global__ void uniDist(
	float *d_a,
	hiprandState_t *states,
	unsigned int size)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < size)
	{
		d_a[i] = hiprand_uniform(&states[i]);
	}
}


__global__ void normalDist(
	float *d_a,
	hiprandState_t *states,
	unsigned int size)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < size)
	{
		d_a[i] = hiprand_normal(&states[i]);
		d_a[i] = curand_
	}
}


int main()
{
	hiprandState_t *states;
	hipMalloc((void**)&states, N*sizeof(hiprandState_t));

	initialize<< <1, numThreads >> >(time(NULL), states, N);

	float h_a[N], h_b[N];
	float *d_a, *d_b;

	hipMalloc((void**)&d_a, N*sizeof(float));
	hipMalloc((void**)&d_b, N*sizeof(float));

	uniDist<<<1,numThreads>>>(
		d_a,
		states,
		N);

	normalDist << <1, numThreads >> >(
		d_b,
		states,
		N);

	hipMemcpy(h_a, d_a, N*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_b, d_b, N*sizeof(float), hipMemcpyDeviceToHost);

	std::cout << std::setw(10);
	std::cout << std::setprecision(4);
	for (int i = 0; i < N; i++)
	{
		std::cout << h_a[i] << " , " << h_b[i] << std::endl;
	}

	hipFree(d_a); hipFree(d_b); hipFree(states);

    return 0;
}
