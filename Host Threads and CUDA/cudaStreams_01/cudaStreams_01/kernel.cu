#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>


unsigned int N = 1000000;
unsigned int numThreads = 512;


__global__ void addArray(
	float *d_a,
	float *d_b,
	float *d_c,
	int size)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i >= size)
	{
		return;
	}
	d_c[i] = d_a[i] + d_b[i];
}



int main()
{
	hipStream_t stream1, stream2, stream3;
	hipEvent_t event1, event2;
	hipEventCreate(&event1); hipEventCreate(&event2);
	hipStreamCreate(&stream1); hipStreamCreate(&stream2); hipStreamCreate(&stream3);

	hipError_t cudaStatus;

	float *h_a, *d_a;
	float *h_b, *d_b;
	float *h_c, *d_c;
	float *h_d, *d_d;
	float *h_e, *d_e;

	cudaStatus = hipHostMalloc(&h_a, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "1\n", cudaStatus);
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_a, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "2\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipHostMalloc(&h_b, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "3\n", cudaStatus);
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_b, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "4\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipHostMalloc(&h_c, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "5\n", cudaStatus);
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_c, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "6\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipHostMalloc(&h_d, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "7\n", cudaStatus);
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_d, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "8\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipHostMalloc(&h_e, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "9\n", cudaStatus);
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_e, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "10\n", cudaStatus);
		goto Error;
	}


	for (int i = 0; i < N; i++)
	{
		h_a[i] = float(i);
		h_b[i] = float(i) ;
		h_c[i] = float(i) ;
	}

	//for (int i = 0; i < N; i++)
	//{
	//	std::cout << h_a[i] << " ";
	//}

	cudaStatus = hipMemcpyAsync(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice, stream1);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "11\n", cudaStatus);
		goto Error;
	}
	cudaStatus = hipMemcpyAsync(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice, stream2);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "12\n", cudaStatus);
		goto Error;
	}
	cudaStatus = hipMemcpyAsync(d_c, h_c, N * sizeof(float), hipMemcpyHostToDevice, stream3);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "13\n", cudaStatus);
		goto Error;
	}


	addArray << < (N + numThreads - 1) / numThreads, numThreads, 0, stream1 >> >(d_a, d_a, d_d, N);

	addArray << < (N + numThreads - 1) / numThreads, numThreads, 0, stream2 >> >(d_b, d_b, d_e, N);


	cudaStatus = hipMemcpyAsync(h_d, d_d, N*sizeof(float), hipMemcpyDeviceToHost, stream1);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "14\n", cudaStatus);
		goto Error;
	}
	hipEventRecord(event1, stream1);
	cudaStatus = hipMemcpyAsync(h_e, d_e, N*sizeof(float), hipMemcpyDeviceToHost, stream2);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "15\n", cudaStatus);
		goto Error;
	}
	hipEventRecord(event1, stream2);
	//hipDeviceSynchronize();
	hipEventSynchronize(event1);
	hipEventSynchronize(event2);

	//std::cout << "2222" << std::endl;
	//for (int i = 0; i < N; i++) {
	//	std::cout << *(h_d + i) << " ";
	//}
	//std::cout << "3333" << std::endl;
	//for (int i = 0; i < N; i++) {
	//	std::cout << *(h_e + i) << " ";
	//}

	hipFree(d_a); hipFree(d_b); hipFree(d_c); hipFree(d_d); hipFree(d_e);
	hipHostFree(h_a); hipHostFree(h_b); hipHostFree(h_c); hipHostFree(h_d); hipHostFree(h_e);
	hipStreamDestroy(stream1); hipStreamDestroy(stream2); hipStreamDestroy(stream3);
	hipEventDestroy(event1); hipEventDestroy(event2);
	hipDeviceReset();

	char temp;
	std::cin >> temp;

	return 0;

Error: 
	{
	hipDeviceReset();
	char temp;
	std::cin >> temp;
	return 0;
	}
	
}
