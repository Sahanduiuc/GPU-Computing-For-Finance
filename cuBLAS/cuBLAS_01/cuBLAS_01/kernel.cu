
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>
const int n = 6;

int main()
{
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	int j;
	float *x;

	x = new float[n];
	for (j = 0; j < n; j++)
	{
		x[j] = float(j);
	}

	float *d_x;
	hipMalloc((void**)&d_x, n*sizeof(float));
	stat = hipblasCreate(&handle);
	stat = hipblasSetVector(n, sizeof(float), x, 1, d_x, 1);
	int result;

	stat = hipblasIsamax(handle, n, d_x, 1, &result);

	std::cout << "The Largest Value is : " << x[result-1] << std::endl;

	hipFree(d_x); 

	hipblasDestroy(handle);

	delete[] x;
	char temp;
	std::cin >> temp;
    return 0;
}
