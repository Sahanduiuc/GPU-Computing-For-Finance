
#include "hip/hip_runtime.h"

#include "hipblas.h"

#include <stdio.h>
#include <iostream>

#define IDX2C(i,j,ld) (((j)*(ld)) + (i))

const int m = 2;
const int n = 3;



int main()
{

	std::cout << "Performing y = alpha * A * x + beta*y " << std::endl;
	std::cout << "Where alpha, beta are scalars" << std::endl;
	std::cout << "x,y are vectors, and A is a matrix" << std::endl;
	std::cout << std::endl;

	hipblasHandle_t handle;
	
	float *a = new float[m*n];
	float *x = new float[n];
	float *y = new float[m];

	int ind = 1;
	for (int j = 0; j < n; j++)
	{
		for (int i = 0; i < m; i++)
		{
			a[IDX2C(i, j, m)] = float(ind++);
		}
	}


	for (int i = 0; i < m; i++)
	{
		for (int j = 0; j < n; j++)
		{
			std::cout << a[IDX2C(i, j, m)] << " ";
		}
		std::cout << std::endl;
	}

	std::cout << std::endl;

	for (int i = 0; i < n; i++)
	{
		x[i] = 1.0f;
	}
	for (int j = 0; j < m; j++)
	{
		y[j] = 0.0f;
	}


	float *d_a, *d_x, *d_y;

	hipMalloc((void**)&d_a, m*n*sizeof(float));
	hipMalloc((void**)&d_x, n*sizeof(float));
	hipMalloc((void**)&d_y, m*sizeof(float));

	hipblasCreate(&handle);
	hipblasSetMatrix(m, n, sizeof(float), a, m, d_a, m);

	hipblasSetVector(n, sizeof(float), x, 1, d_x, 1);
	hipblasSetVector(m, sizeof(float), y, 1, d_y, 1);

	float alpha = 1.0f, beta = 1.0f;

	hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &alpha, d_a, m, d_x, 1, &beta, d_y, 1);

	hipblasGetVector(m, sizeof(float), d_y, 1, y, 1);

	for (int i = 0; i < m; i++)
	{
		std::cout << y[i] << std::endl;
	}

	hipFree(d_a); hipFree(d_x); hipFree(d_y);
	hipblasDestroy(handle);
	delete[] a; delete[] x; delete[] y;
	char temp;
	std::cin >> temp;

    return 0;
}
