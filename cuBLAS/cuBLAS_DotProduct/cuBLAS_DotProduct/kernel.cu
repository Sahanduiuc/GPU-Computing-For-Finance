
#include "hip/hip_runtime.h"

#include "hipblas.h"

#include <stdio.h>
#include <iostream>


const int n = 6;


int main()
{
	hipblasHandle_t handle;
	
	float *x = new float[n];
	float *y = new float[n];


	for (int i = 0; i < n; i++)
	{
		x[i] = float(i);
		y[i] = float(i);
	}


	float *d_x, *d_y;

	hipMalloc((void**)&d_x, n*sizeof(float));
	hipMalloc((void**)&d_y, n*sizeof(float));

	hipblasCreate(&handle);

	hipblasSetVector(n, sizeof(float), x, 2, d_x, 2);
	hipblasSetVector(n, sizeof(float), y, 1, d_y, 1);

	float result;

	hipblasSdot(handle, n, d_x, 1, d_y, 1, &result);
	hipMemcpy(x, d_x, n * sizeof(float), hipMemcpyDeviceToHost);
	
	std::cout << "The Dot Product of x and y is :" << result << std::endl;
	for (int i = 0; i < n; i++) {
		std::cout << *(x+i) << std::endl;
	}
	

	hipFree(d_x); hipFree(d_y);
	hipblasDestroy(handle);
	delete[] x; delete[] y;
	char temp;
	std::cin >> temp;
	return 0;

    return 0;
}
