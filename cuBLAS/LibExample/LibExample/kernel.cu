#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>
using namespace std;
#define INDEX2F(i,j,ld) (i+j*ld)
const int m = 2;
const int n = 3;

float main()
{
	hipblasHandle_t handle;
	float *a, *x, *y;
	float *d_a, *d_x, *d_y;
	a = new float[m*n];
	x = new float[n];
	y = new float[m];
	hipMalloc(&d_a, m*n * sizeof(float));
	hipMalloc(&d_x, n * sizeof(float));
	hipMalloc(&d_y, m * sizeof(float));
	int ind = 0;
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			a[i + j*m] = ind++;
		}
	}
	for (int i = 0; i < m; i++) {
		y[i] = 0;
	}
	for (int i = 0; i < n; i++) {
		x[i] = 1;
	}
	cout << "A matrix is:" << endl;
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			cout << a[i + j*m]<<" ";
		}
		cout << endl;
	}

	hipblasCreate(&handle);
	hipblasSetMatrix(m, n, sizeof(float), a, m, d_a, m);
	hipblasSetVector(n, sizeof(float), x, 1, d_x, 1);
	hipblasSetVector(m, sizeof(float), y, 1, d_y, 1);
	float alpha, beta;
	alpha = 1.0f;
	beta = 1.0f;
	hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &alpha, d_a, m, d_x, 1,&beta, d_y, 1);
	hipblasGetVector(m, sizeof(float), d_y, 1, y, 1);
	
	for (int i = 0; i < m; i++) {
		cout << y[i] << endl;
	}

	hipFree(d_a);
	hipFree(d_x);
	hipFree(d_y);
	delete[] x;
	delete[] y;
	delete[] a;
	hipblasDestroy(handle);

	char temp;
	cin >> temp;


    return 0;
}

