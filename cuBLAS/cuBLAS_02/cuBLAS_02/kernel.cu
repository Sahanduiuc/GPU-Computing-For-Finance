
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <stdlib.h>

#include "hipblas.h"

const int n = 6;

int main()
{
	hipblasHandle_t handle;
	int j;
	float *x;
	x = new float[n];

	for (j = 0; j < n; j++)
	{
		x[j] = float(j);
	}

	float *d_x;
	hipMalloc((void**)&d_x, n*sizeof(float));

	hipblasCreate(&handle);
	hipblasSetVector(n, sizeof(float), x, 1, d_x, 1);
	float result;

	hipblasSasum(handle, n, d_x, 1, &result);
	
	std::cout << "The Sum of the Absolute Value of the Elements Is: " << result << std::endl;

	hipFree(d_x);
	hipblasDestroy(handle);
	delete[] x;

    return 0;
}
