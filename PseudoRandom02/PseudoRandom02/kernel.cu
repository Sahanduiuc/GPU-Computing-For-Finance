#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#define N 50000
#define numThreads 512


__global__ void init(
	unsigned int seed, 
	hiprandState_t *states,
	unsigned int size)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < size)
	{
		hiprand_init(
			seed,
			i,
			0,
			&states[i]);
	}
}


__global__ void GeometricBrownianMotion(
	float *d_a,
	float mu,
	float sigma,
	float dt,
	hiprandState_t *states,
	unsigned int size)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < size)
	{
		d_a[i] += d_a[i] * ( (dt*mu) + (sigma*sqrt(dt)*hiprand_normal(&states[i])));
	}
}


int main()
{
	float mu = .01;
	float sigma = 0.5;
	float dt = 1/252.0f;

	float h_a[N];
	float *d_a;

	for (int i = 0; i < N; i++)
	{
		h_a[i] = 100.0f;
	}
	hipMalloc((void**)&d_a, N*sizeof(float));
	hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);

	hiprandState_t *states;
	hipMalloc((void**)&states, N * sizeof(hiprandState_t));
	init << <(N + numThreads - 1)/numThreads, numThreads >> >(time(NULL), states, N);

	for (int t = 0; t < 252; t++)
	{
		GeometricBrownianMotion << < (N + numThreads - 1) / numThreads, numThreads >> >(
			d_a,
			mu,
			sigma,
			dt,
			states,
			N);
	}

	hipMemcpy(h_a, d_a, N*sizeof(float), hipMemcpyDeviceToHost);

	float avg = 0;

	for (int i = 0; i < N; i++)
	{
		avg += h_a[i];
	}

	avg /= float(N);

	std::cout << "The Average Value Is: " << avg << std::endl;

	hipFree(d_a); hipFree(states);

    return 0;
}
