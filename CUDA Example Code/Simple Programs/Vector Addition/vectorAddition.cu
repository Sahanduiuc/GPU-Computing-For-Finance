
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

#define N 8000
#define numThreads 512

__global__ void addVectors(
	int size,
	float *d_a,
	float *d_b,
	float *d_c)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < size)
	{ 	
		d_c[tid] = d_a[tid] + d_b[tid];
	}
}

int main()
{
	float *h_a,*h_b,*h_c;
	float *d_a, *d_b, *d_c;

	h_a = new float[N];
	h_b = new float[N];
	h_c = new float[N];

	hipMalloc((void**)&d_a, N * sizeof(float));
	hipMalloc((void**)&d_b, N * sizeof(float));
	hipMalloc((void**)&d_c, N * sizeof(float));

	for (int i = 0; i < N; i++)
	{
		h_a[i] = i;
		h_b[i] = i+1;
	}


	hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N*sizeof(float), hipMemcpyHostToDevice);

	addVectors <<<(N + numThreads - 1)/numThreads, numThreads >>>(
		N,
		d_a,
		d_b,
		d_c);

	hipMemcpy(h_c, d_c, N*sizeof(float), hipMemcpyDeviceToHost);

	std::cout << h_c[0] << std::endl;

	delete[] h_a; delete[] h_b; delete[] h_c;
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	return 0;
}