
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

#define N 256

__global__ void addOne(
	float *d_a)
{
	int tid = threadIdx.x;

	d_a[tid] += 1;
}

int main()
{
	float *h_a;
	float *d_a;

	h_a = new float[N];

	hipMalloc((void**)&d_a, N * sizeof(float));

	for (int i = 0; i < N; i++)
	{
		h_a[i] = i;
	}

	std::cout << h_a[0] << std::endl;

	hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);

	addOne <<<1, N >>>(d_a);

	hipMemcpy(h_a, d_a, N*sizeof(float), hipMemcpyDeviceToHost);

	std::cout << h_a[0] << std::endl;

	delete[] h_a;
	hipFree(d_a);

	return 0;
}