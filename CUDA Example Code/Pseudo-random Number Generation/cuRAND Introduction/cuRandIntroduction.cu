#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 128

__global__ void initilize(
	unsigned int seed,
	hiprandState_t *states)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	hiprand_init(
		seed,
		tid,
		0,
		&states[tid]);
}


__global__ void uniformRandom(
	hiprandState_t *states,
	float *d_values)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	d_values[tid] = hiprand_uniform(&states[tid]);
}


int main()
{

	hiprandState_t *d_states;
	hipMalloc((void**)&d_states, N * sizeof(hiprandState_t));

	initilize << <1, N >> >(time(0), d_states);

	float *h_values;
	float *d_values;

	h_values = new float[N];
	
	hipMalloc((void**)&d_values, N * sizeof(float));

	uniformRandom << <1, N >> >(
		d_states,
		d_values);

	hipMemcpy(h_values, d_values, N * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++)
	{
		std::cout << h_values[i] << std::endl;
	}

	delete[] h_values;
	hipFree(d_states);
	hipFree(d_values);

	return 0;
}